
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void safty(int *a, int N,int arg)
{
 int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(arg ==0)
  {
    if(i < N)
    {
        a[i] = 1;
    }
  }
  else
  {
    if(i <N)
    {
    a[i]*=2;
    printf("%d",a[i]);
    }
  }
}

int main()
{
    int *a;
    size_t size = 100;
    int N = size*sizeof(int);
    int thread = 32;
    int block = (N-1)/thread +1;
    hipMallocManaged(&a,N);
    safty<<<block,thread>>>(a,N,0);
    safty<<<block,thread>>>(a,N,1);
    hipDeviceSynchronize();
    hipFree(a);

}
