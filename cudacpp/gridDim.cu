
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>
__global__ void _array(int *a, int N)
{
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride = gridDim.x*blockDim.x;

    for(int i = index; i <N; i+=stride)
    {
            a[i] = 1;   
    }
}


int main(void)
{

    int n = 10000;
    size_t size = sizeof(int)*n;
    int _thread = 32;
    int _block = 100;//(n-1)/_thread + 1;
    int *a;
    hipMallocManaged(&a,size);
    _array<<<_block,_thread>>>(a,size);
    hipDeviceSynchronize();
    for(int i = 0; i<n;i++)
    {
        assert(a[i] == 1);
    }
    printf("success");
    hipFree(a);

}


