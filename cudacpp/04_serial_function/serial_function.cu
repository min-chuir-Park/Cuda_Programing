
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void serial()
{
    printf("%d\n",threadIdx.x);
}


int main()
{
    serial<<<1,10>>>();
    hipDeviceSynchronize();
}
