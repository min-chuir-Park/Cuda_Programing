

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void end()
{
    if(threadIdx.x==31 && blockIdx.x == 1023)
    {
        printf("end\n");
     }
 }

int main()
{
     end<<<1024,32>>>();
     hipDeviceSynchronize();
}     
